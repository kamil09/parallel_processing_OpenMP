#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include ""

#include <stdio.h>
#include <ctime>
#include <stdlib.h>

const int arraySize = 2048*4;
const int block_size = 1024;
hipError_t sumWithCuda(float *c, float *a, unsigned int size, int type);

__global__ void sumKernelStr1(float *c, float *a){
	__shared__ float sdata[arraySize];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	sdata[tid] = a[i];
	__syncthreads();
	for (unsigned int odstep = 1; odstep < blockDim.x; odstep *= 2){
		if (tid %(2*odstep) == 0) sdata[tid] += sdata[tid + odstep];
		__syncthreads();
	}
	if (tid == 0) c[blockIdx.x] = sdata[0];
}

__global__ void sumKernelStr2(float *c, float*a) {
	__shared__ float sdata[arraySize];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	sdata[tid] = a[i];
	__syncthreads();

	for (unsigned int odstep = 1; odstep < blockDim.x; odstep *= 2) {
		int index = 2 * odstep*tid;
		if (index < blockDim.x) sdata[index] += sdata[index + odstep];
		__syncthreads();
	}

	if (tid == 0) c[blockIdx.x] = sdata[0];
}

__global__ void sumKernelStr3(float *c, float *a) {
	__shared__ float sdata[arraySize];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	sdata[tid] = a[i];
	__syncthreads();

	for (unsigned int odstep = blockDim.x / 2; odstep>0; odstep/=2) {
		if (tid < odstep) sdata[tid] += sdata[tid + odstep];
		
		__syncthreads();
	}


	if (tid == 0) c[blockIdx.x] = sdata[0];
}

int main()
{
	srand(time(NULL));
	float a[arraySize];
	for (int i = 0; i < arraySize; i++) a[i] = (float)(rand() % 20);
    float c[1] = { 0 };

    // Sum vector parallel.
	int type = 3;
    hipError_t cudaStatus = sumWithCuda(c, a, arraySize, type);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
		getchar();
        return 1;
    }
	//for (int i = 0; i < arraySize; i++) printf("+%f", a[i]);
    printf("=%f\n",c[0]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
		getchar();
        return 1;
    }

	getchar();
    return 0;
}

// Helper function for using CUDA to sum vector in parallel.
hipError_t sumWithCuda(float *c, float *a, unsigned int size, int type)
{
    float *dev_a = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	hipDeviceProp_t deviceProp;

	cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
	if (deviceProp.computeMode == hipComputeModeProhibited){
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		getchar();
		exit(EXIT_SUCCESS);
	}
	if (cudaStatus != hipSuccess) printf("hipGetDeviceProperties returned error code %d, line(%d)\n", cudaStatus, __LINE__);
	else printf("GPU Device %d: \"%s\" with compute capability %d.%d MP:%d TH_MUL:%d TH:%d WARP:%d\n\n", 0, 
		deviceProp.name, deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount, deviceProp.maxThreadsPerMultiProcessor, deviceProp.maxThreadsPerBlock, deviceProp.warpSize);
	

    // Allocate GPU buffers for 2 vectors (1 input, 1 output).
    cudaStatus = hipMalloc((void**)&dev_c, 1 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	int threads = size;
	if(size>block_size) threads = block_size;
	//printf("%d\n",threads);
	dim3 grid(size / threads);
	// Launch a kernel on the GPU with one thread for each element.

    if(type==1)sumKernelStr1<<<grid, threads >>>(dev_c, dev_a);
	if (type == 2)sumKernelStr2 << <grid, threads >> >(dev_c, dev_a);
	if (type == 3)sumKernelStr3<< <grid, threads >> >(dev_c, dev_a);


    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, 1 * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    
return cudaStatus;
}
