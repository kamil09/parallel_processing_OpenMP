#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include <hip/device_functions.h>
#include ""

#include <stdio.h>
#include <ctime>
#include <stdlib.h>

const int arraySize = 2048*4;
const int block_size = 1024;
hipError_t sumWithCuda(float *c, float *a, unsigned int size, int type);

__global__ void sumKernelStr1(float *c, float *a){
	__shared__ float sdata[block_size];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	sdata[tid] = a[i];
	__syncthreads();
	for (unsigned int odstep = 1; odstep < blockDim.x; odstep *= 2){
		if (tid %(2*odstep) == 0) sdata[tid] += sdata[tid + odstep];
		__syncthreads();
	}
	if (tid == 0) c[blockIdx.x] = sdata[0];
}

__global__ void sumKernelStr2(float *c, float*a) {
	__shared__ float sdata[block_size];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	sdata[tid] = a[i];
	__syncthreads();

	for (unsigned int odstep = 1; odstep < blockDim.x; odstep *= 2) {
		int index = 2 * odstep*tid;
		if (index < blockDim.x) sdata[index] += sdata[index + odstep];
		__syncthreads();
	}

	if (tid == 0) c[blockIdx.x] = sdata[0];
}

__global__ void sumKernelStr3(float *c, float *a) {
	__shared__ float sdata[arraySize];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	sdata[tid] = a[i];
	__syncthreads();

	for (unsigned int odstep = blockDim.x / 2; odstep > 0; odstep /= 2) {
		if (tid < odstep) sdata[tid] += sdata[tid + odstep];

		__syncthreads();
	}
	if (tid == 0) c[blockIdx.x] = sdata[0];
}

int main()
{
	srand(time(NULL));
	float a[arraySize];
	for (int i = 0; i < arraySize; i++) a[i] = (float)(rand() % 20);
    float c[1] = { 0 };

    // Sum vector parallel.
	int type = 3;
    hipError_t cudaStatus = sumWithCuda(c, a, arraySize, type);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
		getchar();
        return 1;
    }
	//for (int i = 0; i < arraySize; i++) printf("+%f", a[i]);
    printf("=%f\n",c[0]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
		getchar();
        return 1;
    }

	getchar();
    return 0;
}

// Helper function for using CUDA to sum vector in parallel.
hipError_t sumWithCuda(float *c, float *a, unsigned int size, int type)
{
    float *dev_a = 0;
    float *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
	hipDeviceProp_t deviceProp;

	cudaStatus = hipGetDeviceProperties(&deviceProp, 0);
	if (deviceProp.computeMode == hipComputeModeProhibited){
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
		getchar();
		exit(EXIT_SUCCESS);
	}
	if (cudaStatus != hipSuccess) printf("hipGetDeviceProperties returned error code %d, line(%d)\n", cudaStatus, __LINE__);
	else printf("GPU Device %d: \"%s\" with compute capability %d.%d MP:%d TH_MUL:%d TH:%d WARP:%d\n\n", 0, 
		deviceProp.name, deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount, deviceProp.maxThreadsPerMultiProcessor, deviceProp.maxThreadsPerBlock, deviceProp.warpSize);
	

    // Allocate GPU buffers for 2 vectors (1 input, 1 output).
    cudaStatus = hipMalloc((void**)&dev_c, 1 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	int threads = size;
	if(size>block_size) threads = block_size;
	//printf("%d\n",threads);
	dim3 grid(size / threads);
	// Launch a kernel on the GPU with one thread for each element.

	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	hipEvent_t stop;
	if ((cudaStatus = hipEventCreate(&start)) != hipSuccess){
		fprintf(stderr, "Failed to create start event (error code %s)!\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);}
	if ((cudaStatus = hipEventCreate(&stop)) != hipSuccess){
		fprintf(stderr, "Failed to create stop event (error code %s)!\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);}
	if ((cudaStatus = hipEventRecord(start, NULL)) != hipSuccess){
		fprintf(stderr, "Failed to record start event (error code %s)!\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);
	}
	int iter = 30000;
	for (int i = 0; i < iter; i++) {
		if (type == 1)sumKernelStr1 << <grid, threads >> > (dev_c, dev_a);
		if (type == 2)sumKernelStr2 << <grid, threads >> > (dev_c, dev_a);
		if (type == 3)sumKernelStr3 << <grid, threads >> > (dev_c, dev_a);
	}

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
   
	if ((cudaStatus = hipEventRecord(stop, NULL)) != hipSuccess){
		fprintf(stderr, "Failed to record stop event (error code %s)!\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);}
	if ((cudaStatus = hipEventSynchronize(stop)) != hipSuccess){
		fprintf(stderr, "Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);}
	float msecTotal = 0.0f;
	if ((cudaStatus = hipEventElapsedTime(&msecTotal, start, stop)) != hipSuccess){
		fprintf(stderr, "Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(cudaStatus));
		exit(EXIT_FAILURE);}

	// Compute and print the performance
	float msecPerVectorSum = msecTotal / iter;
	double flopsPeVectorSum = size;
	double gigaFlops = (flopsPeVectorSum * 1.0e-9f) / (msecPerVectorSum / 1000.0f);
	printf(
		"Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
		gigaFlops,
		msecPerVectorSum,
		flopsPeVectorSum,
		threads);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, 1 * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    
return cudaStatus;
}
